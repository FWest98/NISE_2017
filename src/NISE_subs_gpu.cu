#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "NISE_subs_gpu.cuh"
#include "util/cudaheader.cuh"

// Index triangular matrix
// Put in the .h file to allow external referencing
inline int Sindex(int a, int b, int N) { // inline to make it quicker
    int ind;
    if (a > b) {
        //ind=a+N*b-(b*(b+1)/2);
        ind = a + b * ((N << 1) - b - 1) / 2;
    }
    else {
        //ind=b+N*a-(a*(a+1)/2);
        ind = b + a * ((N << 1) - a - 1) / 2;
    }
    return ind;
}

/* Calculate occupancy */

/* Propagate doubles using diagonal vs. coupling sparce algorithm */
// Kernels

__global__ void expDiag(float *re, float *im, int n, int f, float *H0) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > n) return;
    re[index] = cos(0.5f * H0[index] * f);
    im[index] = -sin(0.5f * H0[index] * f);
}

__global__ void multCVec(float *resR, float *resI, float *xR, float *xI, float *yR, float *yI, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > n) return;
    resR[index] = xR[index] * yR[index] - xI[index] * yI[index];
    resI[index] = xR[index] * yR[index] + xI[index] * yI[index];
}

// Code
void propagate_vec_coupling_S_doubles_GPU(t_non* non, float* Hamiltonian_i, float* cr, float* ci, int m, float* Anh) {
    int N = non->singles;
    int N2 = N * (N + 1) / 2;
    const float f = non->deltat * icm2ifs * twoPi / m;
    float* H0 = (float *) calloc(N2, sizeof(float));
    float* H1 = (float *) calloc(N * N / 2, sizeof(float));
    int* col = (int *) calloc(N * N / 2, sizeof(int));
    int* row = (int *) calloc(N * N / 2, sizeof(int));
    float* re_U = (float *) calloc(N2, sizeof(float));
    float* im_U = (float *) calloc(N2, sizeof(float));
    float* ocr = (float *) calloc(N2, sizeof(float));
    float* oci = (float *) calloc(N2, sizeof(float));

    /* Build Hamiltonians H0 (diagonal) and H1 (coupling) */
    for (int a = 0; a < N; a++) {
        const int indexa = Sindex(a, a, N);
        for (int b = a; b < N; b++) {
            int index = Sindex(a, b, N);
            H0[index] = Hamiltonian_i[indexa] + Hamiltonian_i[Sindex(b, b, N)]; // Diagonal
            if (a == b) {
                if (non->anharmonicity == 0) {
                    H0[index] -= Anh[a];
                }
                else {
                    H0[index] -= non->anharmonicity;
                }
            }
        }
    }

    /* Build Hamiltonian H1 (coupling) */
    int kmax = 0;
    for (int a = 0; a < N; a++) {
        for (int b = a + 1; b < N; b++) {
            int index = b + a * ((N << 1) - a - 1) / 2; // Part of Sindex, but b > a is always true here

            if (fabsf(Hamiltonian_i[index]) > non->couplingcut) {
                H1[kmax] = Hamiltonian_i[index];
                col[kmax] = a, row[kmax] = b;
                kmax++;
            }
        }
    }

    // Move data to GPU
    float* re_U_G, * im_U_G, * cr_G, * ci_G, * ocr_G, * oci_G, * H0_G;
    hipMalloc(&re_U_G, N2 * sizeof(float)); hipMemcpy(re_U_G, re_U, N2 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&im_U_G, N2 * sizeof(float)); hipMemcpy(im_U_G, im_U, N2 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&cr_G, N2 * sizeof(float)); hipMemcpy(cr_G, cr, N2 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&ci_G, N2 * sizeof(float)); hipMemcpy(ci_G, ci, N2 * sizeof(float), hipMemcpyHostToDevice); 
    hipMalloc(&ocr_G, N2 * sizeof(float)); hipMemcpy(ocr_G, ocr, N2 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&oci_G, N2 * sizeof(float)); hipMemcpy(oci_G, oci, N2 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&H0_G, N2 * sizeof(float)); hipMemcpy(H0_G, H0, N2 * sizeof(float), hipMemcpyHostToDevice);

    int gridSize = (N2 + 256 - 1) / 256;

    /* Exponentiate diagonal [U=exp(-i/2h H0 dt)] */
    expDiag KERNEL_ARG2(gridSize, 256) (re_U_G, im_U_G, N2, f, H0_G);
    /*for (int a = 0; a < N2; a++) {
        re_U[a] = cosf(0.5f * H0[a] * f);
        im_U[a] = -sinf(0.5f * H0[a] * f);
    }*/

    hipMemcpy(re_U, re_U_G, N2 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(im_U, im_U_G, N2 * sizeof(float), hipMemcpyDeviceToHost);


    for (int i = 0; i < m; i++) {

        /* Multiply on vector first time */
        for (int a = 0; a < N2; a++) {
            ocr[a] = cr[a] * re_U[a] - ci[a] * im_U[a];
            oci[a] = cr[a] * im_U[a] + ci[a] * re_U[a];
        }

        /* Account for couplings */
        /* Loop over couplings */
        for (int k = 0; k < kmax; k++) {
            int a = col[k];
            int b = row[k];
            float J = H1[k] * f;

            /* Loop over wave functions <ca|Hab|cb> and <cb|Hba|ca> */
            // TODO speedup
            for (int c = 0; c < N; c++) {
                float si = (c == a || c == b) ? -sinf(J * sqrt2) : -sinf(J);

                float co = sqrtf(1 - si * si);
                int index1 = Sindex(a, c, N), index2 = Sindex(c, b, N);
                float cr1 = co * ocr[index1] - si * oci[index2];
                float ci1 = co * oci[index1] + si * ocr[index2];
                float cr2 = co * ocr[index2] - si * oci[index1];
                float ci2 = co * oci[index2] + si * ocr[index1];
                ocr[index1] = cr1, oci[index1] = ci1, ocr[index2] = cr2, oci[index2] = ci2;
            }
        }

        /* Multiply on vector second time */
        for (int a = 0; a < N2; a++) {
            cr[a] = ocr[a] * re_U[a] - oci[a] * im_U[a];
            ci[a] = ocr[a] * im_U[a] + oci[a] * re_U[a];
        }
    }
    free(ocr), free(oci), free(re_U), free(im_U), free(H1), free(H0);
    free(col), free(row);
}
